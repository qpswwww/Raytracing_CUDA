#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include "cuda_error_handle.cuh"
#include "random.cuh"
#include "vec3.cuh"
#include "ray.cuh"
#include "world.cuh"
#include "hittable.cuh"
#include "camera.cuh"
#include "bvh.cuh"
#include "obj_parser.cuh"
#include "FileReader.cuh"

__device__ vec3 color(const ray& r, hittable **world, hiprandState *pixel_random_seed) {
	ray cur_ray = r;
	vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);
	for (int i = 0; i < 50; i++) {
		hit_record rec;
		aabb tmp;
		//bool xxx=((bvh_node*)(*world))->test(tmp);
		//((bvh_node*)(*world))->bounding_box(0.0f,1.0f,tmp);
		//((bvh_node*)(*world))->hit(cur_ray, 0.001f, FLT_MAX, rec);
		
		if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
			ray scattered;
			vec3 attenuation;
			if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, pixel_random_seed)) {
				cur_attenuation *= attenuation;
				cur_ray = scattered;
			}
			else {
				return vec3(0.0, 0.0, 0.0);
			}
		}
		else {
			vec3 unit_direction = unit_vector(cur_ray.direction());
			float t = 0.5f*(unit_direction.y() + 1.0f);
			vec3 c = (1.0f - t)*vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
			return cur_attenuation * c;
		}
	}
	return vec3(0.0, 0.0, 0.0); // exceeded recursion
}

__global__ void render(vec3 *fb, int max_x, int max_y, int ns,camera *cam,hittable **world,hiprandState *random_seed) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = i * max_y + j;
	vec3 col(0, 0, 0);
	hiprandState pixel_random_seed = random_seed[pixel_index];
	//Montocaro random sampling
	for (int s = 0; s < ns; s++) {
		float u = float(i + hiprand_uniform(&pixel_random_seed))/float(max_x);
		float v = float(j + hiprand_uniform(&pixel_random_seed)) / float(max_y);
		ray r = cam->get_ray(u, v,&pixel_random_seed); // , pixel_random_seed);
		col += color(r, world, &pixel_random_seed);
	}
	random_seed[pixel_index] = pixel_random_seed;
	col /= float(ns);
	col[0] = sqrt(col[0]);
	col[1] = sqrt(col[1]);
	col[2] = sqrt(col[2]);
	fb[pixel_index] = col;
}

void GetImage(vec3* fb,int nx,int ny, int ns,char* save_dir,dim3 blocks, dim3 threads, camera *cam, hittable **world,hiprandState *random_seed) {
	FILE *stream1;
	freopen_s(&stream1, save_dir, "w", stdout);
	if (stream1 == NULL) {
		exit(9);
	}
	render<<<blocks, threads>>>(fb, nx, ny, ns, cam, world, random_seed);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	vec3 *fb_host;
	fb_host=(vec3*)malloc(nx*ny * sizeof(vec3));
	checkCudaErrors(hipMemcpy(fb_host, fb, nx*ny * sizeof(vec3),hipMemcpyDeviceToHost));

	std::cout << "P3\n" << nx << " " << ny << "\n255\n";
	for (int j = ny - 1; j >= 0; j--) {
		for (int i = 0; i < nx; i++) {
			int pixel_idx = i*ny+j;
			float r = fb_host[pixel_idx][0];
			float g = fb_host[pixel_idx][1];
			float b = fb_host[pixel_idx][2];
			int ir = int(255.99*r);
			int ig = int(255.99*g);
			int ib = int(255.99*b);
			std::cout << ir << " " << ig << " " << ib << std::endl;
		}
	}
	fclose(stream1);
}

__global__ void visit2(hittable** obj_list,int list_size) {
	hittable *p = (triangle*)obj_list[1];
	material* mp = p->mat_ptr;
	((dielectric*)mp)->ref_idx += 0.01;
}

int main() {
	char dir[200];
	scanf("%s", dir);
	int dx, dy, tx=8, ty=8, ns;
	camera *cam_host;
	
	hittable** obj_list_host;// = (hittable**)malloc((100000) * sizeof(hittable*));
	material** mat_list_host;

	int obj_list_size = 0, mat_list_size = 0;
	FileReader::readfile_to_render(dir,dx,dy,ns, cam_host,obj_list_host,obj_list_size,mat_list_host,mat_list_size);
	std::cerr << "dx=" << dx << " / dy=" << dy << " / tx=" << tx << " / ty=" << ty << " / ns=" << ns << std::endl;

	dim3 blocks(dx / tx + 1, dy / ty + 1), threads(tx, ty);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	//Allocate the frame buffer
	vec3 *fb; //frame buffer
	//checkCudaErrors(hipMallocManaged((void**)&fb, dx*dy * sizeof(vec3)));
	checkCudaErrors(hipMalloc((void**)&fb, dx*dy * sizeof(vec3)));

	//checkCudaErrors(hipGetLastError());
	
	hittable *world_host;
	world_host = new bvh_node(obj_list_host, 0, obj_list_size, 0, 1, 0);

	//Allocate the memory of random seeds
	hiprandState *random_seed;
	//checkCudaErrors(hipMalloc((void**)&random_seed,sizeof(random_seed)));
	checkCudaErrors(hipMalloc((void**)&random_seed, dx*dy * sizeof(hiprandState)));
	init_pixel_random_seed << <blocks, threads >> >(dx, dy, random_seed);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	
	//init_random(dx,dy,random_seed,blocks,threads);
	
	camera* cam_device;
	checkCudaErrors(hipMalloc((void**)&cam_device, sizeof(camera)));
	checkCudaErrors(hipMemcpy(cam_device,cam_host,sizeof(camera),hipMemcpyHostToDevice));

	hittable **world_device;
	checkCudaErrors(hipMalloc((void**)&world_device, sizeof(hittable*)));

	hittable **tmp= new hittable*[1];
	*tmp=world_host->copy_to_gpu();
	checkCudaErrors(hipMemcpy(world_device,tmp, sizeof(hittable*), hipMemcpyHostToDevice));
	//create_world<<<1,1>>>(obj_list,world,cam,dx,dy,random_seed);
	int *count_device,*count_host=new int;
	checkCudaErrors(hipMalloc((void**)&count_device, sizeof(int)));
	visit << <1, 1 >> >((bvh_node**)world_device,count_device);


	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipMemcpy(count_host, count_device, sizeof(int), hipMemcpyDeviceToHost));
	std::cerr << "dbg:count=" << *count_host << std::endl;
	
	//create_world << <1, 1 >> >(world_device, world_device, cam, dx, dy, random_seed);
	//checkCudaErrors(hipGetLastError());
	//checkCudaErrors(hipDeviceSynchronize());

	GetImage(fb,dx,dy,ns,"E://code//Raytracing_CUDA//output.ppm",blocks,threads,cam_device,world_device,random_seed);
	/*
	free_world << <1, 1 >> > (obj_list_device, world_device, cam);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipFree(cam));
	checkCudaErrors(hipFree(world_device));
	//checkCudaErrors(hipFree(obj_list_device));
	checkCudaErrors(hipFree(random_seed));
	checkCudaErrors(hipFree(fb));

	hipDeviceReset();
	*/
	std::cerr << "OKOK" << endl;
	checkCudaErrors(hipFree(cam_device));
	checkCudaErrors(hipFree(random_seed));
	checkCudaErrors(hipFree(fb));
	exit(0);
	return 0;
}